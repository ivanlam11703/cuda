
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int NUM_SAMPLES = 10000000; // 10 million
const int NUM_THREADS = 1000;

__global__ void compute_histogram(int dim, int* histogram, int* size, hiprandState* states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64() + idx, idx, 0, &states[idx]);

    float sum = 0;
    for (int j = 0; j < dim; j++) {
        float val = hiprand_uniform(&states[idx]) * 2 - 1;
        sum += val * val;
    }
    if (sqrt(sum) <= 1.0) {
        float dist_to_surface = 1 - sqrt(sum);
        atomicAdd(&histogram[int(dist_to_surface / 0.01)], 1);
        atomicAdd(size, 1);
    }
}


int main() {
    hiprandState* states;
    hipMalloc((void**)&states, NUM_SAMPLES * sizeof(hiprandState));

    for (int dim = 2; dim <= 16; dim++) {
        int* histogram;
        int* size;

        hipMallocManaged(&histogram, 100 * sizeof(int));
        hipMemset(histogram, 0, 100 * sizeof(int));

        hipMallocManaged(&size, sizeof(int));
        hipMemset(size, 0, sizeof(int));

        compute_histogram<<<(NUM_SAMPLES / NUM_THREADS), NUM_THREADS>>>(dim, histogram, size, states);

        hipDeviceSynchronize();

        std::cout << "Histogram for D = " << dim << ":\n";
        for (int i = 0; i < 100; i++) {
            std::cout << i * 0.01 << " - " << (i + 1) * 0.01 << " : " << (double)histogram[i] / *size << "\n";
        }
        std::cout << "\n";

        hipFree(histogram);
        hipFree(size);
    }

    hipFree(states);

    return 0;
}
